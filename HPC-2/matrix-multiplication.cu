
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <stdexcept>
#include <vector> 
#include <cstdlib>

#define r1 20
#define c1 20
#define r2 20
#define c2 20


__global__ void matmul(int* a,int* b, int* c){
		
	int x = blockIdx.x;
	int y = blockIdx.y;

	c[c2 * y + x] = 0;
	for(int k = 0; k < c1; k++)
		c[c2 * y + x] += (a[c1 * y + k] * b[c2 * k + x]);

}
int main(void){

	srand(time(0));
	int a[r1][c1];
	int b[r2][c2];
	int c[r1][c2];
	

	for(int i = 0; i < r1; i++)
		for(int j = 0; j < c1; j++)
			a[i][j] = rand();

	for(int i = 0; i < r2; i++)
		for(int j = 0; j < c2; j++)
			b[i][j] = rand();

	int *p ,*q, *r;

	hipMalloc((void**)&p,r1 * c1 * sizeof(int));
	hipMalloc((void**)&q,r2 * c2 * sizeof(int));
	hipMalloc((void**)&r,r1 * c2 * sizeof(int));

	hipMemcpy(p,a,r1 * c1 * sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(q,b,r1 * c1 * sizeof(int),hipMemcpyHostToDevice);

	dim3 grid(c2,r1); // cols * rows


	hipEvent_t start,stop;
	float elapsed;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start,0);

	matmul<<<grid,1>>>(p,q,r);

	hipDeviceSynchronize();
	hipMemcpy(c,r, r1 * c2 * sizeof(int),hipMemcpyDeviceToHost);
	hipEventRecord(stop,0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed,start,stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);

	std::cout << "Elapsed Time: " << elapsed  << "ms" << std::endl; 

	hipFree(p);
	hipFree(q);
	hipFree(r);
	return 0;
}
