
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>
#include <stdexcept>
#include <vector> 
#include <cstdlib>

#define n 20

__global__ void add(int *x,int *y, int *z){

  int id=blockIdx.x; 
  z[id]=x[id]+y[id];
}

int main(){

    srand(time(0));
    int a[n], b[n], c[n];

    int *d,*e,*f;
    
    for(int i=0;i<n;i++){
        a[i] = rand();
        b[i] = rand();
    }

    hipMalloc((void **)&d,n*sizeof(int)); 
    hipMalloc((void **)&e,n*sizeof(int));
    hipMalloc((void **)&f,n*sizeof(int));

    hipMemcpy(d,a,n*sizeof(int),hipMemcpyHostToDevice);    
    hipMemcpy(e,b,n*sizeof(int),hipMemcpyHostToDevice); 
 
    hipEvent_t start,stop;
    float elapsed;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start,0);

    add<<<n,1>>>(d,e,f); 

    hipDeviceSynchronize();
    hipMemcpy(c,f,n*sizeof(int),hipMemcpyDeviceToHost); 
    hipEventRecord(stop,0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed,start,stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "Elapsed Time: " << elapsed  << "ms" << std::endl;
    
    hipFree(d); 
    hipFree(e);
    hipFree(f);

    return 0;
}

